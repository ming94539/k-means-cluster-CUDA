#include "hip/hip_runtime.h"
// Imports
#include <algorithm>
#include <cfloat>

#include <chrono>
#include <fstream>


#include <iostream>
#include <random>

#include <sstream>
#include <stdexcept>
#include <vector>

//Distance function to calculate distance etween two points (2 Dim. Data)
__device__ float
2d_distance(float x_a, float y_a, float x_b, float y_b) {
  std::cout << x_a<< std::endl;
  std::cout << y_a<< std::endl;
  std::cout << x_b<< std::endl;
  std::cout << y_b<< std::endl;
  return (y_a-y_b)*(y_a- y_b)+(x_a-x_b)*(x_a- x_b);
}
// Kernel to assign each data to its cluster
__global__ void clusters_assignments(const float* dataX,const float* dataY,int size,const float*  x_avg,const float* y_avg,
float*  totalX,
float*  totalY,int k,
int* counts) {
   __shared__ float centroids[]; //using shared memory to store the averages
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= size) return; //out of scope
  if (threadIdx.x < k) {
    centroidsthreadIdx.x] = x_avg[threadIdx.x];
    centroids[k + threadIdx.x] = y_avg[threadIdx.x];
  }
  __syncthreads();
  // Make global loads once.
  const float x = dataX[index];
  const float y = dataY[index];
  float best_distance = FLT_MAX; //Longest possible distance
  int best_cluster = 0;
  for (int cluster = 0; cluster < k; ++cluster) {
    const float distance = 2d_distance(x,
 y,shared_means[cluster], shared_means[k + cluster]);
    if (distance < best_distance) { //Keep finding the shortest distance
       best_distance = distance;
        best_cluster = cluster; //assign to its cluster
    }
  }
  atomicAdd(&totalX[best_cluster], x); //Need to be optimized into reduction, but atomic operation for now
  atomicAdd(&totalY[best_cluster], y); 
  atomicAdd(&counts[best_cluster], 1);
}
//Recalculate the new centroids and reset eveyrhting back to 0 
__global__ void compute_new_means_and_reset(float* meansX,float* meansY,float* Totalx,
float* Totaly, int*   counts) {
  const int cluster= threadIdx.x; 
  const int count =max(1, counts[cluster]);
  meansX[cluster] =new_sum_x[cluster]/count;
  meansY[cluster] =new_sum_y[cluster]/ count;
  Totalx[cluster]=0;
  Totaly[cluster]= 0;
  counts[cluster] =0;
}
//Create a Data class to help organize and generalize all the vectors while also providing
//generalizable functions
struct Points {
  Points(int size) : size(size), bytes(size * sizeof(float)) {
    hipMalloc(&x,bytes); //allocate data onto GPU
    hipMalloc(&y,bytes);
    hipMemset(x,0, bytes);
    hipMemset(y,0, bytes);}
  Points(int size, std::vector<float>& h_x, std::vector<float>& h_y)
  : size(size), bytes(size * sizeof(float)) {
    hipMalloc(&x,bytes);
    hipMalloc(&y,bytes);
    hipMemcpy(x, h_x.data(), bytes, hipMemcpyHostToDevice); //transfer data to gpu
    hipMemcpy(y,h_y.data(), bytes, hipMemcpyHostToDevice);
}
  ~Points(){ //to free data later on
    hipFree(x);
  hipFree(y);
  } 
  float* x{nullptr};
  float* y{nullptr};
  int size{0};
  int bytes{0} ;
};
int main(int argc, const char* argv[]) {
  if(argc < 3){ // parse csv file 
    std::cerr << "Didn't input right amount of arguments correctly"<< std::endl;
    std::exit(EXIT_FAILURE);
}
  const auto k = std::atoi(argv[2]); //k clusters
  const auto number_of_iterations = std::atoi(argv[2]); //Number of iterations
  std::vector<float> h_x; //assuming 2 dimensional data, so vector containing x axis
  std::vector<float> h_y; //vector containing y axis
  std::ifstream stream(argv[1]); //parse the csv file
  std::string line;
  while (std::getline(stream, line)) { //goes through the csv file line by line
    std::istringstream line_stream(line);
    std::cout << line << std::endl;
    float x, y;
    uint16_t label;
    line_stream >> x >> y >> label; //assume the csv file is in format: x y label
    h_x.push_back(x); 
    h_y.push_back(y);
  }
  const size_t number_of_elements = h_x.size(); //N size of dataset
  Points d_data(number_of_elements, h_x, h_y); 
  Points d_means(k, h_x, h_y); 
  Points d_sums(k); 
  int* d_counts;
  hipMalloc(&d_counts, k * sizeof(int)); //allocate the space for the centroids
  hipMemset(d_counts, 0, k * sizeof(int)); 
  const int threads = 1024;
  const int blocks = (number_of_elements + threads - 1) / threads;
  const int shared_memory = d_means.bytes * 2;
  std::cerr << "Processing " << number_of_elements << " points on " << blocks
            << " blocks x " << threads << " threads" << std::endl;
  const auto start = std::chrono::high_resolution_clock::now(); 
  for (size_t iteration = 0; iteration < number_of_iterations; ++iteration) {
    cluster_assignments<<<blocks, threads, shared_memory>>>(d_data.x,d_data.y,d_data.size,d_means.x,d_means.y,
d_sums.x,d_sums.y,k,d_counts); //first kernel to assign clusters
    hipDeviceSynchronize(); //make sure all operations have been completed first
    compute_new_means_and_reset<<<1, k>>>(d_means.x,
d_means.y, d_sums.x,d_sums.y,d_counts); //the recalaculation operation in seperate kernel
    hipDeviceSynchronize(); //make sure all operations have been completed first
  }
  const auto end = std::chrono::high_resolution_clock::now(); //timer ends
  const auto duration = std::chrono::duration_cast<std::chrono::duration<float>>(end - start);
  std::cerr << "Took: " << duration.count() << "s" << std::endl; //calculate duration
  hipFree(d_counts); // free memory
  std::vector<float> mean_x(k, 0);
  std::vector<float> mean_y(k, 0);
  hipMemcpy(mean_x.data(), d_means.x, d_means.bytes, hipMemcpyDeviceToHost); //copy data points x axis back to host
  // copy data points y axis back to host
  hipMemcpy(mean_y.data(), d_means.y, d_means.bytes, hipMemcpyDeviceToHost);
  //print out all the clusters
  for (size_t cluster = 0; cluster < 1; ++cluster) {
    std::cout << mean_x[cluster] << " " << mean_y[cluster] << std::endl;
  }
}










// References: 
// https://github.com/src-d/kmcuda
// https://stanford.edu/~cpiech/cs221/handouts/kmeans.html
// http://www.goldsborough.me/
// https://www.geeksforgeeks.org/k-means-clustering-introduction/
// https://reasonabledeviations.com/
// https://rosettacode.org/
// https://github.com/robertmartin8/udemyML/blob/master/06_clustering/Mall_Customers.csv